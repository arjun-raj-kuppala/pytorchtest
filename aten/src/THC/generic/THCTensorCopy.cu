
#include <hip/hip_runtime.h>
#ifndef THC_GENERIC_FILE
#define THC_GENERIC_FILE "THC/generic/THCTensorCopy.cu"
#else

void THCTensor_(copy)(THCState* state, THCTensor* dst, THCTensor* src) {
  if (dst == src) return;
  at::Tensor dst_wrap = THTensor_wrap(dst);
  at::Tensor src_wrap = THTensor_wrap(src);
  at::native::copy_(dst_wrap, src_wrap);
}

template <>
THCTensor *THCTensor_newClone<scalar_t>(THCState *state, THCTensor *self) {
  THCTensor* tensor =
      // THCTensor_new(state, THTensor_getStoragePtr(self)->dtype());
      THCTensor_new(state, self->dtype());
  THCTensor_resizeAs(state, tensor, self);
  at::Tensor tensor_wrap = THTensor_wrap(tensor);
  at::Tensor self_wrap = THTensor_wrap(self);
  at::native::copy_(tensor_wrap, self_wrap);
  return tensor;
}

template <>
THCTensor *THCTensor_newContiguous<scalar_t>(THCState *state, THCTensor *self)
{
  if(!self->is_contiguous()) {
    return THCTensor_newClone<scalar_t>(state, self);
  } else {
    THCTensor_retain(state, self);
    return self;
  }
}


template <>
void THCTensor_freeCopyTo<scalar_t>(THCState *state, THCTensor *self, THCTensor *dst) {
  if(self != dst) {
    at::Tensor dst_wrap = THTensor_wrap(dst);
    at::Tensor self_wrap = THTensor_wrap(self);
    at::native::copy_(dst_wrap, self_wrap);
  }

  THCTensor_free(state, self);
}

#endif
